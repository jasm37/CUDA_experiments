// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}


__global__
void sumFunc(float *a, float *b, float *c, int n)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i<n) c[i] = a[i] + b[i];
}


int main(int argc, char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 20;
    float *a = new float[n];
    float *b = new float[n];
    float *c = new float[n];
    for(int i=0; i<n; i++)
    {
        a[i] = i;
        b[i] = (i%5)+1;
        c[i] = 0;
    }

    // CPU computation
    for(int i=0; i<n; i++) c[i] = a[i] + b[i];

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;
    // init c
    for(int i=0; i<n; i++) c[i] = 0;
    
    float *d_a, *d_b, *d_c = NULL;
    size_t nbytes = size_t(n)*sizeof(float);

    // Initialize stuff
    dim3 block = dim3(128,1,1);
    dim3 grid = dim3((n + block.x -1) / block.x, 1, 1);

    hipMalloc(&d_a, nbytes);
    hipMalloc(&d_b, nbytes);
    hipMalloc(&d_c, nbytes);
    CUDA_CHECK;

    hipMemcpy( d_a, a, nbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_b, b, nbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_c, c, nbytes, hipMemcpyHostToDevice );
    CUDA_CHECK;

    //Call function
    sumFunc<<<grid, block>>>(d_a, d_b, d_c, n);

    hipMemcpy( a, d_a, nbytes, hipMemcpyDeviceToHost );
    hipMemcpy( b, d_b, nbytes, hipMemcpyDeviceToHost );
    hipMemcpy( c, d_c, nbytes, hipMemcpyDeviceToHost );
    CUDA_CHECK;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    CUDA_CHECK;



    // GPU computation
    // ###
    // ### TODO: Implement the array addition on the GPU, store the result in "c"
    // ###
    // ### Notes:
    // ### 1. Remember to free all GPU arrays after the computation
    // ### 2. Always use the macro CUDA_CHECK after each CUDA call, e.g. "cudaMalloc(...); CUDA_CHECK;"
    // ###    For convenience this macro is defined directly in this file, later we will only include "helper.h"
    


    // print result
    cout << "GPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
    delete[] b;
    delete[] c;
}



