
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

int main()
{
    float num[100000];
    float *d_num;
    int n = 100000;
    size_t nbytes = n * sizeof(float);
    // fill array :
    for(int i = 0; i < n; i++) num[i] = i;

    hipMalloc(&d_num, nbytes);
    hipMemcpy(d_num, num, nbytes, hipMemcpyHostToDevice);

    // Start handle to use cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Variable where to store sum:
    float sum_num = 0;
    
    hipblasSasum(handle, n, d_num, 1, &sum_num);
    std::cout << "Sum is "<< sum_num << std::endl;

    hipblasDestroy(handle);
    hipFree(d_num);

     
}
