#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
using namespace std;

// uncomment to use the camera
//#define CAMERA


__global__ void histogram_global(float *d_imgIn, int *d_hist, int w, int h, int nc)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    // SImply count values into histogram array.
    // Here I assume that we want only one histogram and we just sum up over the channels
    // instead of making one histogram per channel
	if (x < w && y < h && z < nc)
	{
		int ind = x + w*y + w*h*z; 
		int index = d_imgIn[ind]*255.f;
		atomicAdd(&d_hist[index], 1);
	}
}

__global__ void histogram_shared(float *d_imgIn, int *d_hist, int dimx, int dimy, int nc)
{
    int num_bin = 256;
    __shared__ int hist[256];

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int k = threadIdx.z + blockDim.z * blockIdx.z;


    if(i < dimx && j < dimy && k < nc)
    { 
        // First set to zero to start counting
        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) 
        {
            for(int it = 0; it < num_bin; it++)
                hist[it] = 0;
        }
        
        __syncthreads();

        // Count values from all pixels/threads(per block)
        int ind = i + j * dimx + dimx*dimy*k;
        int bin_label = d_imgIn[ind]*255.f;
        atomicAdd(&hist[bin_label], 1);

        __syncthreads();

        // Sum all the shared memories over every blocks
        if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
        {
            for(int it = 0; it < num_bin; it++)
                atomicAdd(&d_hist[it], hist[it]);
        }
        __syncthreads();
    } 
}



int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];

    int hist_size = 256;
    int *hist_global = new int[hist_size];
    int *hist_shared = new int[hist_size];
    
	//allocate memory on device
	float *d_imgIn;
	int *d_hist;
	int imgSize = (size_t)w*h*nc;
	
	hipMalloc(&d_imgIn, imgSize*sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_hist, hist_size*sizeof(int)); CUDA_CHECK;

    dim3 block = dim3(16, 4, 4);
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, (nc + block.z - 1)/block.z);
    



    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

        /////////////////////////
        // Exercise starts here
        ////////////////////////

	//copy host memory to device
	hipMemcpy(d_imgIn, imgIn, imgSize*sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
    hipMemset(d_hist, 0, hist_size*sizeof(int));
    

    Timer timer;
    
    timer.start();
    // call global memory histogram
    histogram_global<<<grid,block>>> (d_imgIn, d_hist, w, h, nc);
    timer.end();  float t = timer.get(); 
    cout << "Global time: " << t*1000 << " ms" << endl;
    hipMemcpy(hist_global, d_hist, hist_size * sizeof(int), hipMemcpyDeviceToHost); CUDA_CHECK;

    timer.start();
    //Call shared memory histogram
    histogram_shared<<<grid,block>>> (d_imgIn, d_hist, w, h, nc);
    timer.end();  t = timer.get(); 
    cout << "Shared time: " << t*1000 << " ms" << endl;
    hipMemcpy(hist_shared, d_hist, hist_size * sizeof(int), hipMemcpyDeviceToHost); CUDA_CHECK;
    
    // Plot histograms
    showHistogram256("Histogram global", hist_global, 500, 100);
    showHistogram256("Histogram shared", hist_shared, 500, 300);
	

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



