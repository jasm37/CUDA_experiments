// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}

__global__
void squareFunc(float *a, int n)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i<n) a[i] = a[i] * a[i];
}


int main(int argc,char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 10;
    float *a = new float[n];
    for(int i=0; i<n; i++) a[i] = i;

    // CPU computation
    for(int i=0; i<n; i++)
    {
        float val = a[i];
        val = val*val;
        a[i] = val;
    }

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;
    


    // GPU computation
    // reinit data
    for(int i=0; i<n; i++) a[i] = i;

    // Initialize blocks and array
    dim3 block = dim3(128,1,1);
    dim3 grid = dim3((n + block.x -1) / block.x, 1, 1);

    // Set size and memory in GPU
    float *d_a = NULL;
    size_t nbytes = size_t(n)*sizeof(float);

    hipMalloc(&d_a, nbytes);
    CUDA_CHECK;

    hipMemcpy( d_a, a, nbytes, hipMemcpyHostToDevice );
    CUDA_CHECK;

    //Call function
    squareFunc<<<grid, block>>>(d_a, n);

    hipMemcpy( a, d_a, nbytes, hipMemcpyDeviceToHost );
    CUDA_CHECK;

    hipFree(d_a);
    CUDA_CHECK;
    
    // ###
    // ### TODO: Implement the "square array" operation on the GPU and store the result in "a"
    // ###
    // ### Notes:
    // ### 1. Remember to free all GPU arrays after the computation
    // ### 2. Always use the macro CUDA_CHECK after each CUDA call, e.g. "cudaMalloc(...); CUDA_CHECK;"
    // ###    For convenience this macro is defined directly in this file, later we will only include "helper.h"


    // print result
    cout << "GPU:" << endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
}



