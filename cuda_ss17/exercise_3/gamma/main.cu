#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
#include <math.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

__global__
void gammaCorr(float *a, float gamma, int n)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if (i<n) a[i] = powf(a[i],gamma);
}


int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;
    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 2;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    float gamma = 1;
    getParam("gamma", gamma, argc, argv);
    cout << "gamma: " << gamma << endl;

    // ### Define your own parameters here as needed    

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);



    ////////////////////////
    // Exercise starts here
    ////////////////////////    
    int size_elem = w*h*nc;
    Timer timer; timer.start();
    int nComb = 5;
    int x_arr[] = {32, 64, 96, 128, 256, 512};
    int y_arr[] = {8, 4, 2, 2, 1, 1 };
    ///////////
    // CPU code
    ///////////
    for(int j = 0; j < repeats; j++)
        for(int i = 0; i<size_elem; i++)
        {
                imgOut[i] = pow(imgIn[i], gamma);
        }

    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "Avg. CPU time: " << t*1000 / repeats << " ms" << endl;


    ///////////
    //GPU code
    ///////////
    Timer mTimer;
    int dim_x, dim_y;
    // nComb is number of combinations in dimx and dimy
    for(int run = 0; run<nComb; run++)
    {
        mTimer.start();
        dim_x = x_arr[run];
        dim_y = y_arr[run];

        // Set size and memory in GPU
        size_elem = w*h*nc;
        float *d_imgOut = NULL;
        size_t nbytes = size_t(size_elem)*sizeof(float);

        // Initialize stuff
        dim3 block = dim3(dim_x,dim_y,1);
        dim3 grid = dim3((size_elem + block.x -1) / block.x, 1, 1);

        hipMalloc(&d_imgOut, nbytes);CUDA_CHECK;

        hipMemcpy( d_imgOut, imgIn, nbytes, hipMemcpyHostToDevice );CUDA_CHECK;

        //Call function
        
        gammaCorr<<<grid, block>>>(d_imgOut, gamma, size_elem);
        
        hipMemcpy( imgOut, d_imgOut, nbytes, hipMemcpyDeviceToHost );CUDA_CHECK;

        hipFree(d_imgOut);CUDA_CHECK;

        mTimer.end();  t = mTimer.get();  // elapsed time in seconds
        cout << "GPU time for (dim_x="<< dim_x <<", dim_y="<< dim_y <<") is " << t*1000 << " ms" << endl;
    }





    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output in GPU", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



