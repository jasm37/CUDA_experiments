#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#define _USE_MATH_DEFINES
#include "helper.h"
#include <iostream>
#include <math.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA


__device__
void clampGPU(int &i, int &j, int dimx, int dimy)
{
    if(i<0) i=0;
    else if(i>dimx-1) i= dimx-1;
    if(j<0) j=0;
    else if(j>dimy-1) j= dimy-1;
}

__global__
void div(float *div_vec, float *dx_a, float *dy_a, float *dxx_a, float *dyy_a, int dimx, int dimy, int n)
{
    int ind = threadIdx.x + blockDim.x*blockIdx.x;
    int sub = ind %(dimx*dimy);
    if(ind < dimx*dimy)
    {
        if(sub%dimx == 0) dxx_a[ind]=(dx_a[ind]-dx_a[ind-1]);
        if( sub >= dimx ) dyy_a[ind]=(dy_a[ind]-dy_a[ind-dimx]);
        div_vec[ind] = dxx_a[ind] + dyy_a[ind];
    }
}

__global__
void d_plus_rot(float *a, float *d_grad, float *dx_a, float *dy_a, int dimx, int dimy, int nc)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    int k = threadIdx.z + blockDim.z*blockIdx.z;
    // Assumes that dx_a and dy_a are initialized as zero arrays
    float cte = 1.f/32.f;
    float temp_dx = 0, temp_dy = 0;
    if(i < dimx && j < dimy && k < nc)
    {
        int temp_pi = i+1, temp_pj=j+1, temp_ni=i-1, temp_nj=j-1;
        if(i==0) temp_ni = 0;
        else if(i==dimx-1) temp_pi = dimx-1;
        if(j==0) temp_nj=0;
        else if(j==dimy-1) temp_pj = dimy-1;

        temp_dx=cte*(
                    3*a[temp_pi + dimx*(temp_pj) + dimx*dimy*k] +
                    10*a[temp_pi + dimx*j + dimx*dimy*k] +
                    3*a[temp_pi + dimx*temp_nj + dimx*dimy*k]
                    -(3*a[temp_ni + dimx*temp_pj + dimx*dimy*k] +
                    10*a[temp_ni + dimx*j + dimx*dimy*k] +
                    3*a[temp_ni + dimx*temp_nj + dimx*dimy*k]));
        temp_dy=cte*(
                    3*a[temp_pi + dimx*(temp_pj) + dimx*dimy*k] +
                    10*a[i + dimx*temp_pj + dimx*dimy*k] +
                    3*a[temp_ni + dimx*temp_pj + dimx*dimy*k]
                    -(3*a[temp_pi + dimx*temp_nj + dimx*dimy*k] +
                    10*a[i + dimx*temp_nj + dimx*dimy*k] +
                    3*a[temp_ni + dimx*temp_nj + dimx*dimy*k]));
        dx_a[i+dimx*j+dimx*dimy*k] = temp_dx;
        dy_a[i+dimx*j+dimx*dimy*k] = temp_dy;
        d_grad[i+dimx*j+dimx*dimy*k] = temp_dx + temp_dy;
    }
}

__global__
void compute_M(float *coord_1, float *coord_2, float *coord_3, float *dx_a, float *dy_a, int dimx, int dimy, int nc)
{
    // n is the number of channels
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    int k = threadIdx.z + blockDim.z*blockIdx.z;
    int ind, ind2;
    float temp1 = 0, temp2 = 0, temp3 = 0;
    if (i < dimx && j < dimy && k == 0 )
    {
        for (int n = 0; n < nc; n++)
        {
            ind = i+dimx*j+dimx*dimy*n;
            temp1 += dx_a[ind]*dx_a[ind];
            temp2 += dx_a[ind]*dy_a[ind];
            temp3 += dy_a[ind]*dy_a[ind];
        }

        ind2 = i+dimx*j;
        coord_1[ind2] = temp1;
        coord_2[ind2] = temp2;
        coord_3[ind2] = temp3;
    }
}

__global__
void do_GPUconvolution(float *conv, float *ker, float *a, int r, int dimx, int dimy, int nc)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int n = threadIdx.z + blockDim.z * blockIdx.z;
    int drad = 2*r+1; // double radius
    int ni, nj;
    double temp;
    if( i < dimx && j <dimy && n<nc )
    {
        temp = 0;
        for(int ir = 0; ir<drad; ir++)
            for(int jr = 0; jr<drad; jr++)
            {
                ni = i-ir+r;
                nj = j-jr+r;
                clampGPU(ni, nj, dimx, dimy);
                temp += ker[ir + drad*jr]*a[ni + dimx*nj + dimx*dimy*n];
            }
        conv[i + dimx*j + dimx*dimy*n] = temp;
    }

}


void kernel_comp(float *ker, float std, int radius)
{
    float coeff = 1 / (2 * M_PI * std * std);
    float arg;
    int rad = radius*2+1;
    int in, jn;
    float sum= 0;

    for(int i=0; i<rad; i++)
        for(int j=0; j<rad; j++)
        {   in = i-radius; jn = j-radius;
            arg = -( in*in + jn*jn ) / (2*std*std);
            ker[i + rad*j] = coeff * expf(arg);
            sum += ker[i + rad*j];
        }

    for(int i=0; i<rad; i++)
        for(int j=0; j<rad; j++)
            ker[i + rad*j] /= sum;

}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;
    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif

    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;

    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    float sigma = 0.3f;
    getParam("sigma", sigma, argc, argv);
    cout << "sigma: " << sigma << endl;

    float scale = 10.f;
    getParam("scale", scale, argc, argv);
    cout << "scale: " << scale << endl;

    // ### Define your own parameters here as needed

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
        cv::VideoCapture camera(0);
        if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
        camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
        camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;

#else

    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }

#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;

    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###

    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    ////////////////////////////
    //Exercise starts here
    ///////////////////////////

    // First kernel
    //float sigma = 0.3;
    int rad_sigma = ceil(3*sigma);
    int double_rad_sigma = 2*rad_sigma+1;
    float *ker_sigma = new float[double_rad_sigma*double_rad_sigma];
    kernel_comp(ker_sigma, sigma, rad_sigma);

    // Second kernel
    float rho = sigma;
    int rad_rho = ceil(3*rho);
    int double_rad_rho = 2*rad_rho+1;
    float *ker_rho = new float[double_rad_rho*double_rad_rho];
    kernel_comp(ker_rho, rho, rad_rho);


    int size_elem = w*h*nc;

    float *d_ker_rho, *d_ker_sigma, *d_conv, *d_imgIn, *d_imgOut, *d_grad, *d_dx_conv, *d_dy_conv, *d_coeff1, *d_coeff2, *d_coeff3, *d_coeff_joint;
    float *dd_coeff1, *dd_coeff2, *dd_coeff3;
    float *m_coord1 = new float[w*h], *m_coord2 = new float[w*h], *m_coord3 = new float[w*h];
    float *dx_a = new float[size_elem];
    float *coeff_joint = new float[4*w*h];
    float *conv = new float[size_elem];

    size_t nbytes = size_t(size_elem)*sizeof(float);

    int dim_x = 16;
    int dim_y = 4;
    int dim_z = 4;
    // Initialize blocks and grid
    dim3 block = dim3(dim_x,dim_y,dim_z);
    dim3 grid = dim3((w + block.x -1) / block.x, (h + block.y -1) / block.y, (nc + block.z -1) / block.z);

    // Memory allocation
    // Implemented two kernels in case the respective sigmas were different
    hipMalloc(&d_ker_sigma, double_rad_sigma*double_rad_sigma*sizeof(float));CUDA_CHECK;
    hipMemcpy( d_ker_sigma, ker_sigma, double_rad_sigma*double_rad_sigma*sizeof(float), hipMemcpyHostToDevice );CUDA_CHECK;
    hipMalloc(&d_ker_rho, double_rad_rho*double_rad_rho*sizeof(float));CUDA_CHECK;
    hipMemcpy( d_ker_rho, ker_rho, double_rad_rho*double_rad_rho*sizeof(float), hipMemcpyHostToDevice );CUDA_CHECK;

    hipMalloc(&d_grad, nbytes);CUDA_CHECK;
    hipMemset(d_grad, 0, nbytes);
    hipMalloc(&d_dx_conv, nbytes);CUDA_CHECK;
    hipMemset(d_dx_conv, 0, nbytes);
    hipMalloc(&d_dy_conv, nbytes);CUDA_CHECK;
    hipMemset(d_dy_conv, 0, nbytes);
    hipMalloc(&d_conv, nbytes);CUDA_CHECK;

    hipMalloc(&d_imgOut, nbytes);CUDA_CHECK;
    hipMemset(d_imgOut, 0, nbytes);
    hipMalloc(&d_imgIn, nbytes);CUDA_CHECK;
    hipMemcpy( d_imgIn, imgIn, nbytes, hipMemcpyHostToDevice );CUDA_CHECK;

    size_t coeff_size = 4*w*h*sizeof(float);
    hipMalloc(&d_coeff_joint, coeff_size);CUDA_CHECK;
    hipMemset(d_coeff_joint, 0, nbytes);

    // d_coeff stores the coefficient of the matrix M
    size_t grid_bytes = size_t(w*h)*sizeof(float);
    hipMalloc(&d_coeff1, grid_bytes);CUDA_CHECK;
    hipMemset(d_coeff1, 0, grid_bytes);
    hipMalloc(&d_coeff2, grid_bytes);CUDA_CHECK;
    hipMemset(d_coeff2, 0, grid_bytes);
    hipMalloc(&d_coeff3, grid_bytes);CUDA_CHECK;
    hipMemset(d_coeff3, 0, grid_bytes);

    // dd_coeff stores the output of the convolution of the coefficnet of M
    hipMalloc(&dd_coeff1, grid_bytes);CUDA_CHECK;
    hipMalloc(&dd_coeff2, grid_bytes);CUDA_CHECK;
    hipMalloc(&dd_coeff3, grid_bytes);CUDA_CHECK;

    //////// Computations ///////////
    Timer timer; timer.start();
    // 1. Compute first convolution : S = G * u
    do_GPUconvolution<<<grid, block>>>(d_conv, d_ker_sigma, d_imgIn, rad_sigma, w, h, nc);

    // 2. Compute rotationally symmetric derivatives
    d_plus_rot<<<grid, block>>>(d_conv, d_grad, d_dx_conv, d_dy_conv, w, h,nc);
    
    // 3. Compute matrix M per pixel
    compute_M<<<grid, block>>>(/*d_coeff_joint,*/ d_coeff1, d_coeff2, d_coeff3, d_dx_conv, d_dy_conv, w, h, nc);
    hipMemcpy( m_coord1, d_coeff1, grid_bytes, hipMemcpyDeviceToHost );CUDA_CHECK;
    hipMemcpy( m_coord2, d_coeff2, grid_bytes, hipMemcpyDeviceToHost );CUDA_CHECK;
    hipMemcpy( m_coord3, d_coeff3, grid_bytes, hipMemcpyDeviceToHost );CUDA_CHECK;

    // 4. Convolution to entries of matrix M : T = G * M
    do_GPUconvolution<<<grid, block>>>(dd_coeff1, d_ker_rho, d_coeff1, rad_rho, w, h, 1);
    hipMemcpy( m_coord1, dd_coeff1, grid_bytes, hipMemcpyDeviceToHost );CUDA_CHECK;

    do_GPUconvolution<<<grid, block>>>(dd_coeff2, d_ker_rho, d_coeff2, rad_rho, w, h, 1);
    hipMemcpy( m_coord2, dd_coeff2, grid_bytes, hipMemcpyDeviceToHost );CUDA_CHECK;

    do_GPUconvolution<<<grid, block>>>(dd_coeff3, d_ker_rho, d_coeff3, rad_rho, w, h, 1);
    hipMemcpy( m_coord3, dd_coeff3, grid_bytes, hipMemcpyDeviceToHost );CUDA_CHECK;

    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "Full procedure time: " << t*1000 << " ms" << endl;

    //float scale = 10.f;
    cv::Mat mCoord1(h,w,CV_32FC1);
    convert_layered_to_mat(mCoord1, m_coord1);
    showImage("C1_out", scale*mCoord1, 100+w+10, 100);
    cv::Mat mCoord2(h,w,CV_32FC1);
    convert_layered_to_mat(mCoord2, m_coord2);
    showImage("C2_out", scale*mCoord2, 100+w+20, 100);
    cv::Mat mCoord3(h,w,CV_32FC1);
    convert_layered_to_mat(mCoord3, m_coord3);
    showImage("C3_out", scale*mCoord3, 100+w+30, 100);

    hipMemcpy( imgOut, d_grad, nbytes, hipMemcpyDeviceToHost );CUDA_CHECK;
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // Free memory
    hipFree(d_imgOut);CUDA_CHECK;
    hipFree(d_ker_rho);CUDA_CHECK;
    hipFree(d_ker_sigma);CUDA_CHECK;
    hipFree(d_conv);CUDA_CHECK;
    hipFree(d_imgIn);CUDA_CHECK;
    hipFree(dd_coeff1);CUDA_CHECK;
    hipFree(dd_coeff2);CUDA_CHECK;
    hipFree(dd_coeff3);CUDA_CHECK;
    hipFree(d_coeff1);CUDA_CHECK;
    hipFree(d_coeff2);CUDA_CHECK;
    hipFree(d_coeff3);CUDA_CHECK;
    hipFree(d_dx_conv);CUDA_CHECK;
    hipFree(d_dy_conv);CUDA_CHECK;

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;
    //delete[] ker;
    //delete[] conv;
    // close all opencv windows
    cvDestroyAllWindows();

    return 0;
}


