#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
#include <math.h>
using namespace std;

// 2. Compute diff coeff
__host__ __device__
float scalar_func(float a, int type, float eps)
{
    float output;
    if(type == 0)
    {
        output = 1.0f;
    }
    else if(type == 1)
    {
        output = 1.0f / (eps>a ? eps : a);
    }
    else if(type == 2)
    {
        output = exp(-a*a / eps) / eps;
    }

    return output;
}

// 3. COmpute derivative and scale it according to diffusivity
__global__
void d_plus(float *a, float *dx_a, float *dy_a, int type, float eps, int dimx, int dimy, int nc)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    int k = threadIdx.z + blockDim.z*blockIdx.z;
    // Assumes that dx_a and dy_a are initialized as zero arrays
    float tempx = 0, tempy = 0;

    float sc, norm=0;
    int ind = 0;
    if(i < dimx && j < dimy && k < nc)
    {
        ind = i + j*dimx + dimx*dimy*k;
        if( i < dimx-1 )  tempx=(a[ind+1]-a[ind]);
        if( j < dimy-1 )  tempy=(a[ind+dimx]-a[ind]);

        norm = sqrtf(tempx*tempx + tempy*tempy);
        sc = scalar_func(norm, type, eps);
        dx_a[ind] = sc*tempx;
        dy_a[ind] = sc*tempy;

    }
}


__global__
void div(float *div_vec, float *dx_a, float *dy_a, int dimx, int dimy, int nc)
{
    // n is the number of channels
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    int k = threadIdx.z + blockDim.z*blockIdx.z;
    // Assumes that dxx_a and dyy_a are initialized as zero arrays
    //int sub = ind %(dimx*dimy);
    float tempx = 0, tempy = 0;
    int ind = 0;
    if(i < dimx && j < dimy && k < nc)
    {
        ind = i + j*dimx + dimx*dimy*k;
        if(i > 0) tempx=(dx_a[ind]-dx_a[ind-1]);
        if(j > 0 ) tempy=(dy_a[ind]-dy_a[ind-dimx]);
        div_vec[ind] = tempx + tempy;
    }
}


__global__
void time_step(float *a , float *div_vec, float tau, int dimx, int dimy, int nc)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int j = threadIdx.y + blockDim.y*blockIdx.y;
    int k = threadIdx.z + blockDim.z*blockIdx.z;
    float un, unp;
    int ind;
    if(i<dimx && j <dimy && k < nc)
    {
        ind = i + j*dimx + k*dimx*dimy;
        un = a[ind];
        unp = un + tau * div_vec[ind];

        a[ind] = unp;
    }
}


int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif

    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;

    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    int nit = 1;
    getParam("nit", nit, argc, argv);
    cout << "number of it: " << nit << endl;

    int type = 0;
    getParam("type", type, argc, argv);
    cout << "type of scalar g: " << nit << endl;

    float eps = 0.01;
    getParam("eps", eps, argc, argv);
    cout << "eps: " << eps << endl;

    float tau = 0.0001;
    getParam("tau", tau, argc, argv);
    cout << "tau: " << tau << endl;

    // ### Define your own parameters here as needed

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
        cv::VideoCapture camera(0);
        if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
        camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
        camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;

#else

    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }

#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;

    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];
    //float *imgOut = new float[(size_t)w*h];

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    //////////////////////
    //Exercise starts here
    /////////////////////
    int size_elem = w*h*nc;
    float *d_imgOut, *dx_a, *dy_a, *d_div_vec, *d_imgIn;
    size_t nbytes = size_t(size_elem)*sizeof(float);

    int dim_x = 32;
    int dim_y = 4;
    int dim_z = 2;
   
    // Initialize block and grid
    dim3 block = dim3(dim_x,dim_y,dim_z);
    dim3 grid = dim3((w + block.x -1) / block.x, (h + block.y -1) / block.y, (nc + block.z -1) / block.z);

    // memory allocation
    hipMalloc(&d_imgOut, nbytes);CUDA_CHECK;
    hipMemset(d_imgOut, 0, nbytes);
    hipMalloc(&d_imgIn, nbytes);CUDA_CHECK;
    hipMemset(d_imgIn, 0, nbytes);
    hipMalloc(&dx_a, nbytes);CUDA_CHECK;
    hipMemset(dx_a, 0, nbytes);
    hipMalloc(&dy_a, nbytes);CUDA_CHECK;
    hipMemset(dy_a, 0, nbytes);
    hipMalloc(&d_div_vec, nbytes);CUDA_CHECK;
    hipMemset(d_div_vec, 0, nbytes);
    hipMemcpy( d_imgIn, imgIn, nbytes, hipMemcpyHostToDevice );CUDA_CHECK;


    Timer mTimer;mTimer.start();
    //Call function: start timestepping!
    for(int it = 0; it < nit; it++)
    {
        // 1.Compute derivatives and scale them with g_hat
        d_plus<<<grid, block>>>(d_imgIn, dx_a, dy_a, type, eps, w, h, nc);
        // 4.Compute divergence of scaled derivative
        div<<<grid, block>>>(d_div_vec, dx_a, dy_a, w, h, nc);
        // 5.Compute next time step of the image
        time_step<<<grid, block>>>(d_imgIn, d_div_vec, tau, w, h, nc);
    }

    // show output image: first convert to interleaved opencv format from the layered raw array
    hipMemcpy( imgOut, d_imgIn, w*h*nc*sizeof(float), hipMemcpyDeviceToHost );CUDA_CHECK;
    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)        
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    hipFree(d_imgOut);CUDA_CHECK;
    hipFree(d_imgIn);CUDA_CHECK;
    hipFree(dx_a);
    hipFree(dy_a);
    hipFree(d_div_vec);


    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



