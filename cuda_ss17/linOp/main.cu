#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
#include <math.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA


__global__
void l2norm(float *a, float *b, int dimx, int dimy, int n)
{
    int ind = threadIdx.x + blockDim.x*blockIdx.x;
    int norm = 0;
    if(ind < dimx*dimy)
    {
        for(int i = 0; i<n; i++)
        {
            norm += pow(a[ind + i*dimx*dimy], 2);
        }
        b[ind] = sqrtf(norm);
    }

}

__global__
void div(float *div_vec, float *dx_a, float *dy_a, float *dxx_a, float *dyy_a, int dimx, int dimy, int n)
{
    // n is the number of channels
    int ind = threadIdx.x + blockDim.x*blockIdx.x;
    // Assumes that dxx_a and dyy_a are initialized as zero arrays
    int sub = ind %(dimx*dimy);
    if(ind < dimx*dimy*n)
    {
        if(sub%dimx == 0) dxx_a[ind]=(dx_a[ind]-dx_a[ind-1]);
        if( sub >= dimx ) dyy_a[ind]=(dy_a[ind]-dy_a[ind-dimx]);
        div_vec[ind] = dxx_a[ind] + dyy_a[ind];
    }
}

__global__
void d_plus(float *a, float *dx_a, float *dy_a, int dimx, int dimy, int n)
{
    // n is the number of channels
    int ind = threadIdx.x + blockDim.x*blockIdx.x;
    // Assumes that dx_a and dy_a are initialized as zero arrays
    int sub = ind %(dimx*dimy);
    if(ind < dimx*dimy*n)
    {
        if( (sub+1)%dimx != 0 ) dx_a[ind]=(a[ind+1]-a[ind]);
        if( sub <= dimy*(dimx-1) ) dy_a[ind]=(a[ind+dimx]-a[ind]);
    }
}



int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *imgOut = new float[(size_t)w*h*mOut.channels()];
    //float *imgOut = new float[(size_t)w*h];




    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);


    //Timer timer; timer.start();
    // ###
    // ###
    // ### TODO: Main computation
    // ###
    // ###
    int size_elem = w*h*nc;
    float *d_imgOut, *dx_a, *dy_a, *dxx_a, *dyy_a, *div_vec;
    float *result = new float[(size_t)w*h];
    size_t nbytes = size_t(size_elem)*sizeof(float);

    int dim_x = 32;
    int dim_y = 8;
    // Initialize stuff
    dim3 block = dim3(dim_x,dim_y,1);
    dim3 grid = dim3((size_elem + block.x -1) / block.x, 1, 1);

    hipMalloc(&d_imgOut, nbytes);CUDA_CHECK;
    hipMemset(d_imgOut, 0, nbytes);
    hipMalloc(&dx_a, nbytes);CUDA_CHECK;
    hipMemset(dx_a, 0, nbytes);
    hipMalloc(&dy_a, nbytes);CUDA_CHECK;
    hipMemset(dy_a, 0, nbytes);
    hipMalloc(&dxx_a, nbytes);CUDA_CHECK;
    hipMemset(dxx_a, 0, nbytes);
    hipMalloc(&dyy_a, nbytes);CUDA_CHECK;
    hipMemset(dyy_a, 0, nbytes);
    hipMalloc(&div_vec, w*h*sizeof(float));CUDA_CHECK;
    hipMemset(div_vec, 0, w*h*sizeof(float));
    //hipMalloc(&result, w*h*sizeof(float));CUDA_CHECK;
    //hipMemset(result, 0, w*h*sizeof(float));CUDA_CHECK;

    hipMemcpy( d_imgOut, imgIn, nbytes, hipMemcpyHostToDevice );CUDA_CHECK;

    //Call function
    Timer mTimer;mTimer.start();
    //float *a, float *dx_a, float *dy_a, int dimx, int dimy, int n)
    // n is the number of channels

    d_plus<<<grid, block>>>(d_imgOut, dx_a, dy_a, w, h, nc);
    div<<<grid, block>>>(div_vec, dx_a, dy_a, dxx_a, dyy_a, w, h, nc);
    CUDA_CHECK;

    //d_plus<<<grid, block>>>(d_imgOut, dx_a, dy_a, w, h, nc);
    //mTimer.end();  t = mTimer.get();  // elapsed time in seconds
    //l2norm<<<grid, block>>>(div_vec, result, w, h, nc);
    hipMemcpy( result, div_vec, w*h*sizeof(float), hipMemcpyDeviceToHost );CUDA_CHECK;
    cv::Mat mRes(h,w,CV_32FC3);
    convert_layered_to_mat(mRes, result);
    //hipMemcpy( imgOut, dx_a, w*h*nc*sizeof(float), hipMemcpyDeviceToHost );CUDA_CHECK;

    hipFree(d_imgOut);CUDA_CHECK;
    hipFree(dx_a);CUDA_CHECK;
    hipFree(dy_a);CUDA_CHECK;
    hipFree(dxx_a);CUDA_CHECK;
    hipFree(dyy_a);CUDA_CHECK;
    hipFree(div_vec);CUDA_CHECK;
    hipFree(result);CUDA_CHECK;
    //timer.end();  float t = timer.get();  // elapsed time in seconds
    //cout << "time: " << t*1000 << " ms" << endl;






    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, imgOut);
    showImage("Output", mRes, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



